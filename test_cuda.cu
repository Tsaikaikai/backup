

#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("Number of CUDA devices: %d\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);
        //printf("Device %d: %s\n", i, deviceProp.name);
    }

    return 0;
}